#include <gtest/gtest.h>
#include "horizontal_path_aggregation.hpp"
#include "path_aggregation_test.hpp"
#include "generator.hpp"
#include "test_utility.hpp"

#include "debug.hpp"

TEST(HorizontalPathAggregationTest, RandomLeft2Right){
	static constexpr size_t width = 631, height = 479, disparity = 128;
	static constexpr unsigned int p1 = 20, p2 = 100;
	const auto left  = generate_random_sequence<sgm::feature_type>(width * height);
	const auto right = generate_random_sequence<sgm::feature_type>(width * height);
	const auto expect = path_aggregation(
		left, right, width, height, disparity, p1, p2, 1, 0);

	const auto d_left = to_device_vector(left);
	const auto d_right = to_device_vector(right);
	thrust::device_vector<sgm::cost_type> d_cost(width * height * disparity);
	sgm::path_aggregation::enqueue_aggregate_left2right_path<disparity>(
		d_cost.data().get(),
		d_left.data().get(),
		d_right.data().get(),
		width, height, p1, p2, 0);
	hipStreamSynchronize(0);

	const auto actual = to_host_vector(d_cost);
	EXPECT_EQ(actual, expect);
	debug_compare(actual.data(), expect.data(), width, height, disparity);
}

TEST(HorizontalPathAggregationTest, RandomRight2Left){
	static constexpr size_t width = 640, height = 480, disparity = 64;
	static constexpr unsigned int p1 = 20, p2 = 40;
	const auto left  = generate_random_sequence<sgm::feature_type>(width * height);
	const auto right = generate_random_sequence<sgm::feature_type>(width * height);
	const auto expect = path_aggregation(
		left, right, width, height, disparity, p1, p2, -1, 0);

	const auto d_left = to_device_vector(left);
	const auto d_right = to_device_vector(right);
	thrust::device_vector<sgm::cost_type> d_cost(width * height * disparity);
	sgm::path_aggregation::enqueue_aggregate_right2left_path<disparity>(
		d_cost.data().get(),
		d_left.data().get(),
		d_right.data().get(),
		width, height, p1, p2, 0);
	hipStreamSynchronize(0);

	const auto actual = to_host_vector(d_cost);
	EXPECT_EQ(actual, expect);
	debug_compare(actual.data(), expect.data(), width, height, disparity);
}
