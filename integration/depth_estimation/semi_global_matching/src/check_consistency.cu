#include "hip/hip_runtime.h"
/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <libsgm.h>
#include "internal.h"

namespace {
	template<typename SRC_T, typename DST_T>
	__global__ void check_consistency_kernel(DST_T* d_leftDisp, const DST_T* d_rightDisp, const SRC_T* d_left, int width, int height, int src_pitch, int dst_pitch, bool subpixel)  {

		const int j = blockIdx.x * blockDim.x + threadIdx.x;
		const int i = blockIdx.y * blockDim.y + threadIdx.y;

		// left-right consistency check, only on leftDisp, but could be done for rightDisp too

		SRC_T mask = d_left[i * src_pitch + j];
		int d = d_leftDisp[i * dst_pitch + j];
		if (subpixel) {
			d >>= sgm::StereoSGM::SUBPIXEL_SHIFT;
		}
		int k = j - d;
		if (mask == 0 || d <= 0 || (k >= 0 && k < width && abs(d_rightDisp[i * dst_pitch + k] - d) > 1)) {
			// masked or left-right inconsistent pixel -> invalid
			d_leftDisp[i * dst_pitch + j] = 0;
		}
	}
}

namespace sgm {
	namespace details {

		void check_consistency(uint8_t* d_left_disp, const uint8_t* d_right_disp, const void* d_src_left, int width, int height, int depth_bits, int src_pitch, int dst_pitch, bool subpixel) {

			const dim3 blocks(width / 16, height / 16);
			const dim3 threads(16, 16);
			if (depth_bits == 16) {
				check_consistency_kernel<uint16_t> << < blocks, threads >> > (d_left_disp, d_right_disp, (uint16_t*)d_src_left, width, height, src_pitch, dst_pitch, subpixel);
			}
			else if (depth_bits == 8) {
				check_consistency_kernel<uint8_t> << < blocks, threads >> > (d_left_disp, d_right_disp, (uint8_t*)d_src_left, width, height, src_pitch, dst_pitch, subpixel);
			}

			CudaKernelCheck();
		}

		void check_consistency(uint16_t* d_left_disp, const uint16_t* d_right_disp, const void* d_src_left, int width, int height, int depth_bits, int src_pitch, int dst_pitch, bool subpixel) {

			const dim3 blocks(width / 16, height / 16);
			const dim3 threads(16, 16);
			if (depth_bits == 16) {
				check_consistency_kernel<uint16_t> << < blocks, threads >> > (d_left_disp, d_right_disp, (uint16_t*)d_src_left, width, height, src_pitch, dst_pitch, subpixel);
			}
			else if (depth_bits == 8) {
				check_consistency_kernel<uint8_t> << < blocks, threads >> > (d_left_disp, d_right_disp, (uint8_t*)d_src_left, width, height, src_pitch, dst_pitch, subpixel);
			}
			
			CudaKernelCheck();	
		}

	}
}
