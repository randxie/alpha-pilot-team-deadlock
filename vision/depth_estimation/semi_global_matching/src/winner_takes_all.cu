#include "hip/hip_runtime.h"
/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <cstdio>
#include <libsgm.h>
#include "winner_takes_all.hpp"
#include "utility.hpp"

namespace sgm {

namespace {

static constexpr unsigned int NUM_PATHS = 8u;

static constexpr unsigned int WARPS_PER_BLOCK = 8u;
static constexpr unsigned int BLOCK_SIZE = WARPS_PER_BLOCK * WARP_SIZE;


__device__ inline void update_top2(uint32_t& v0, uint32_t& v1, uint32_t x){
	const uint32_t y = max(x, v0);
	v0 = min(x, v0);
	v1 = min(y, v1);
}

struct Top2 {
	uint32_t values[2];

	__device__ void initialize(){
		values[0] = 0xffffffffu;
		values[1] = 0xffffffffu;
	}

	__device__ void push(uint32_t x){
		update_top2(values[0], values[1], x);
	}
};

template <unsigned int GROUP_SIZE, unsigned int STEP>
struct subgroup_merge_top2_impl {
	static __device__ Top2 call(Top2 x){
#if CUDA_VERSION >= 9000
		const uint32_t a = __shfl_xor_sync(0xffffffffu, x.values[0], STEP / 2, GROUP_SIZE);
		const uint32_t b = __shfl_xor_sync(0xffffffffu, x.values[1], STEP / 2, GROUP_SIZE);
#else
		const uint32_t a = __shfl_xor(x.values[0], STEP / 2, GROUP_SIZE);
		const uint32_t b = __shfl_xor(x.values[1], STEP / 2, GROUP_SIZE);
#endif
		x.push(a);
		x.push(b);
		return subgroup_merge_top2_impl<GROUP_SIZE, STEP / 2>::call(x);
	}
};

template <unsigned int GROUP_SIZE>
struct subgroup_merge_top2_impl<GROUP_SIZE, 1u> {
	static __device__ Top2 call(Top2 x){
		return x;
	}
};

template <unsigned int GROUP_SIZE>
__device__ inline Top2 subgroup_merge_top2(Top2 x){
	return subgroup_merge_top2_impl<GROUP_SIZE, GROUP_SIZE>::call(x);
}

__device__ inline uint32_t pack_cost_index(uint32_t cost, uint32_t index){
	union {
		uint32_t uint32;
		ushort2 uint16x2;
	} u;
	u.uint16x2.x = static_cast<uint16_t>(index);
	u.uint16x2.y = static_cast<uint16_t>(cost);
	return u.uint32;
}

__device__ uint32_t unpack_cost(uint32_t packed){
	return packed >> 16;
}

__device__ uint32_t unpack_index(uint32_t packed){
	return packed & 0xffffu;
}

using ComputeDisparity = uint32_t(*)(Top2, float, uint16_t*);

template <size_t MAX_DISPARITY>
__device__ inline uint32_t compute_disparity_normal(Top2 t2, float uniqueness, uint16_t* smem = nullptr)
{
	const float cost0 = static_cast<float>(unpack_cost(t2.values[0]));
	const float cost1 = static_cast<float>(unpack_cost(t2.values[1]));
	const int disp0 = static_cast<int>(unpack_index(t2.values[0]));
	const int disp1 = static_cast<int>(unpack_index(t2.values[1]));
	if(cost1 * uniqueness >= cost0){
		return disp0;
	}else if(abs(disp1 - disp0) <= 1){
		return disp0;
	}else{
		return 0;
	}
}

template <size_t MAX_DISPARITY>
__device__ inline uint32_t compute_disparity_subpixel(Top2 t2, float uniqueness, uint16_t* smem)
{
	const float cost0 = static_cast<float>(unpack_cost(t2.values[0]));
	const float cost1 = static_cast<float>(unpack_cost(t2.values[1]));
	const int disp0 = static_cast<int>(unpack_index(t2.values[0]));
	const int disp1 = static_cast<int>(unpack_index(t2.values[1]));
	if(cost1 * uniqueness >= cost0
		|| abs(disp1 - disp0) <= 1){
		int disp = disp0;
		disp <<= sgm::StereoSGM::SUBPIXEL_SHIFT;
		if (disp0 > 0 && disp0 < MAX_DISPARITY - 1) {
			const int numer = smem[disp0 - 1] - smem[disp0 + 1];
			const int denom = smem[disp0 - 1] - 2 * smem[disp0] + smem[disp0 + 1];
			disp += ((numer << sgm::StereoSGM::SUBPIXEL_SHIFT) + denom) / (2 * denom);
		}
		return disp;
	}else{
		return 0;
	}
}


template <unsigned int MAX_DISPARITY, ComputeDisparity compute_disparity = compute_disparity_normal<MAX_DISPARITY>>
__global__ void winner_takes_all_kernel(
	output_type *left_dest,
	output_type *right_dest,
	const cost_type *src,
	int width,
	int height,
	int pitch,
	float uniqueness)
{
	static const unsigned int ACCUMULATION_PER_THREAD = 16u;
	static const unsigned int REDUCTION_PER_THREAD = MAX_DISPARITY / WARP_SIZE;
	static const unsigned int ACCUMULATION_INTERVAL = ACCUMULATION_PER_THREAD / REDUCTION_PER_THREAD;
	static const unsigned int UNROLL_DEPTH = 
		(REDUCTION_PER_THREAD > ACCUMULATION_INTERVAL)
			? REDUCTION_PER_THREAD
			: ACCUMULATION_INTERVAL;

	const unsigned int cost_step = MAX_DISPARITY * width * height;
	const unsigned int warp_id = threadIdx.x / WARP_SIZE;
	const unsigned int lane_id = threadIdx.x % WARP_SIZE;

	const unsigned int y = blockIdx.x * WARPS_PER_BLOCK + warp_id;
	src += y * MAX_DISPARITY * width;
	left_dest  += y * pitch;
	right_dest += y * pitch;

	if(y >= height){
		return;
	}

	__shared__ uint16_t smem_cost_sum[WARPS_PER_BLOCK][ACCUMULATION_INTERVAL][MAX_DISPARITY];

	Top2 right_top2[REDUCTION_PER_THREAD];
	for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
		right_top2[i].initialize();
	}

	for(unsigned int x0 = 0; x0 < width; x0 += UNROLL_DEPTH){
#pragma unroll
		for(unsigned int x1 = 0; x1 < UNROLL_DEPTH; ++x1){
			if(x1 % ACCUMULATION_INTERVAL == 0){
				const unsigned int k = lane_id * ACCUMULATION_PER_THREAD;
				const unsigned int k_hi = k / MAX_DISPARITY;
				const unsigned int k_lo = k % MAX_DISPARITY;
				const unsigned int x = x0 + x1 + k_hi;
				if(x < width){
					const unsigned int offset = x * MAX_DISPARITY + k_lo;
					uint32_t sum[ACCUMULATION_PER_THREAD];
					for(unsigned int i = 0; i < ACCUMULATION_PER_THREAD; ++i){
						sum[i] = 0;
					}
					for(unsigned int p = 0; p < NUM_PATHS; ++p){
						uint32_t load_buffer[ACCUMULATION_PER_THREAD];
						load_uint8_vector<ACCUMULATION_PER_THREAD>(
							load_buffer, &src[p * cost_step + offset]);
						for(unsigned int i = 0; i < ACCUMULATION_PER_THREAD; ++i){
							sum[i] += load_buffer[i];
						}
					}
					store_uint16_vector<ACCUMULATION_PER_THREAD>(
						&smem_cost_sum[warp_id][k_hi][k_lo], sum);
				}
#if CUDA_VERSION >= 9000
				__syncwarp();
#else
				__threadfence_block();
#endif
			}
			const unsigned int x = x0 + x1;
			if(x < width){
				// Load sum of costs
				const unsigned int smem_x = x1 % ACCUMULATION_INTERVAL;
				const unsigned int k0 = lane_id * REDUCTION_PER_THREAD;
				uint32_t local_cost_sum[REDUCTION_PER_THREAD];
				load_uint16_vector<REDUCTION_PER_THREAD>(
					local_cost_sum, &smem_cost_sum[warp_id][smem_x][k0]);
				// Pack sum of costs and dispairty
				uint32_t local_packed_cost[REDUCTION_PER_THREAD];
				for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
					local_packed_cost[i] = pack_cost_index(local_cost_sum[i], k0 + i);
				}
				// Update left
				Top2 left_top2;
				left_top2.initialize();
				for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
					left_top2.push(local_packed_cost[i]);
				}
				left_top2 = subgroup_merge_top2<WARP_SIZE>(left_top2);
				if(lane_id == 0){
					left_dest[x] = compute_disparity(left_top2, uniqueness, smem_cost_sum[warp_id][smem_x]);
				}
				// Update right
#pragma unroll
				for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
					const unsigned int k = lane_id * REDUCTION_PER_THREAD + i;
					const int p = static_cast<int>(((x - k) & ~(MAX_DISPARITY - 1)) + k);
					const unsigned int d = static_cast<unsigned int>(x - p);
#if CUDA_VERSION >= 9000
					const uint32_t recv = __shfl_sync(0xffffffffu,
						local_packed_cost[(REDUCTION_PER_THREAD - i + x1) % REDUCTION_PER_THREAD],
						d / REDUCTION_PER_THREAD,
						WARP_SIZE);
#else
					const uint32_t recv = __shfl(
						local_packed_cost[(REDUCTION_PER_THREAD - i + x1) % REDUCTION_PER_THREAD],
						d / REDUCTION_PER_THREAD,
						WARP_SIZE);
#endif
					right_top2[i].push(recv);
					if(d == MAX_DISPARITY - 1){
						if(0 <= p){
							right_dest[p] = compute_disparity_normal<MAX_DISPARITY>(right_top2[i], uniqueness);
						}
						right_top2[i].initialize();
					}
				}
			}
		}
	}
	for(unsigned int i = 0; i < REDUCTION_PER_THREAD; ++i){
		const unsigned int k = lane_id * REDUCTION_PER_THREAD + i;
		const int p = static_cast<int>(((width - k) & ~(MAX_DISPARITY - 1)) + k);
		if(p < width){
			right_dest[p] = compute_disparity_normal<MAX_DISPARITY>(right_top2[i], uniqueness);
		}
	}
}

template <size_t MAX_DISPARITY>
void enqueue_winner_takes_all(
	output_type *left_dest,
	output_type *right_dest,
	const cost_type *src,
	int width,
	int height,
	int pitch,
	float uniqueness,
	bool subpixel,
	hipStream_t stream)
{
	const int gdim =
		(height + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
	const int bdim = BLOCK_SIZE;
	if (subpixel) {
		winner_takes_all_kernel<MAX_DISPARITY, compute_disparity_subpixel<MAX_DISPARITY>><<<gdim, bdim, 0, stream>>>(
			left_dest, right_dest, src, width, height, pitch, uniqueness);
	} else {
		winner_takes_all_kernel<MAX_DISPARITY, compute_disparity_normal<MAX_DISPARITY>><<<gdim, bdim, 0, stream>>>(
			left_dest, right_dest, src, width, height, pitch, uniqueness);
	}
}

}


template <size_t MAX_DISPARITY>
WinnerTakesAll<MAX_DISPARITY>::WinnerTakesAll()
	: m_left_buffer()
	, m_right_buffer()
{ }

template <size_t MAX_DISPARITY>
void WinnerTakesAll<MAX_DISPARITY>::enqueue(
	const cost_type *src,
	int width,
	int height,
	int pitch,
	float uniqueness,
	bool subpixel,
	hipStream_t stream)
{
	if(m_left_buffer.size() != static_cast<size_t>(pitch * height)){
		m_left_buffer = DeviceBuffer<output_type>(pitch * height);
	}
	if(m_right_buffer.size() != static_cast<size_t>(pitch * height)){
		m_right_buffer = DeviceBuffer<output_type>(pitch * height);
	}
	enqueue_winner_takes_all<MAX_DISPARITY>(
		m_left_buffer.data(),
		m_right_buffer.data(),
		src,
		width,
		height,
		pitch,
		uniqueness,
		subpixel,
		stream);
}

template <size_t MAX_DISPARITY>
void WinnerTakesAll<MAX_DISPARITY>::enqueue(
	output_type* left,
	output_type* right,
	const cost_type *src,
	int width,
	int height,
	int pitch,
	float uniqueness,
	bool subpixel,
	hipStream_t stream)
{
	enqueue_winner_takes_all<MAX_DISPARITY>(
		left,
		right,
		src,
		width,
		height,
		pitch,
		uniqueness,
		subpixel,
		stream);
}


template class WinnerTakesAll< 64>;
template class WinnerTakesAll<128>;

}
